/*
 * Squaring Numbers using CUDA

 * Author: Anirudha Bose <ani07nov@gmail.com>
 * Part of CS344: Intro to Parallel Programming
 * http://github.com/onyb/mooca
 */



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_out, float *d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f;
}

int main(int agrc, char **argv){
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	int i;

	/* Generate the input array on the host */
	float h_in[ARRAY_SIZE];
	
	for(i=0; i<ARRAY_SIZE; i++)
		h_in[i] = float(i);

	float h_out[ARRAY_SIZE];

	/* Declare GPU memory pointers */
	float *d_in;
	float *d_out;

	/* Allocate GPU memory */
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	/* Transfer the array to the GPU */
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	/* Launch the GPU kernel */
	square <<<1, ARRAY_SIZE>>>(d_out, d_in);

	/* Copy back the resulting array to CPU */
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	/* Print out the resulting array */
	for(i=0; i<ARRAY_SIZE; i++){
		printf("%f", h_out[i]);
		printf(((i%4)!=3) ? "\t" : "\n");
	}

	/* Free GPU memory allocation */
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}